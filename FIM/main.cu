#include "hip/hip_runtime.h"
#include <StructuredEikonal.h>
#include <happly.h>
#include "../mesh_functions.h"
#include "../FMM/mesh_functions_fmm.h"
#include <chrono>
#include <cmath>

int main()
{
    happly::PLYData plyIn(R"(C:\Users\Maks\CLionProjects\Agate3DEvolver\AgateContour_0209.ply)");
    std::vector<std::array<double, 3>> vPos = plyIn.getVertexPositions();
    std::vector<std::vector<size_t>> fInd = plyIn.getFaceIndices<size_t>();

    const int max_x = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[0] < b[0]);}))[0]);
    const int max_y = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[1] < b[1]);}))[1]);
    const int max_z = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[2] < b[2]);}))[2]);
    /*const int min_x = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[0] < b[0]);}))[0]);
    const int min_y = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[1] < b[1]);}))[1]);
    const int min_z = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[2] < b[2]);}))[2]);
                                                    */
    const int min_x = 0, min_y = 0, min_z = 0;
    const size_t X = max_x - min_x, Y = max_y - min_y, Z = max_z - min_z;
    auto *mesh = new triangle[fInd.size()];
    for(size_t i = 0; i < fInd.size(); ++i)
    {
        mesh[i][0][0] = vPos[fInd[i][0]][0] - min_x;
        mesh[i][0][1] = vPos[fInd[i][0]][1] - min_y;
        mesh[i][0][2] = vPos[fInd[i][0]][2] - min_z + 0.4;

        mesh[i][1][0] = vPos[fInd[i][1]][0] - min_x;
        mesh[i][1][1] = vPos[fInd[i][1]][1] - min_y;
        mesh[i][1][2] = vPos[fInd[i][1]][2] - min_z + 0.4;

        mesh[i][2][0] = vPos[fInd[i][2]][0] - min_x;
        mesh[i][2][1] = vPos[fInd[i][2]][1] - min_y;
        mesh[i][2][2] = vPos[fInd[i][2]][2] - min_z + 0.4;
    }
    auto start = std::chrono::high_resolution_clock::now();

    auto boundary_indices = find_seeds_fim(mesh, X, Y, Z, fInd.size());

    std::cout << fInd.size() << " triangles loaded from mesh! Starting calculations... \n";

    StructuredEikonal data(false);
    data.setDims(360, 360, 360);
    data.setSeeds(boundary_indices);
    data.solveEikonal();
    auto arrival_times = data.answer_;

    for(int z = 0; z < Z; ++z)
    {
        for(int y = 0; y < Y; ++y)
        {
            for(int x = 0; x < X; ++x)
            {
                if(outside_mesh(mesh, x, y, z, fInd.size(), Y))
                {
                    arrival_times[x][y][z] = -std::fabs(arrival_times[x][y][z]);
                }
                else
                {
                    arrival_times[x][y][z] = std::fabs(arrival_times[x][y][z]);
                }
            }
        }
    }

    delete[] mesh;

    /*
    void StructuredEikonal::setDims(size_t w, size_t h, size_t d);  //set the volume dimensions
    void StructuredEikonal::setMapType(size_t t); //pre-generated speed functions (sphere or egg-carton)
    void StructuredEikonal::setItersPerBlock(size_t t); //set the iterations per block
    void StructuredEikonal::setSpeeds(std::vector<std::vector<std::vector<double> > > speed); //set the voxel speeds
    void StructuredEikonal::setSeeds(std::vector<std::array<size_t, 3> > seeds); //set list of seed voxels
     */
    auto stop = std::chrono::high_resolution_clock::now();


    std::ofstream logfile("log.txt", std::ios::app);
    logfile << "FIM " << std::chrono::duration<double, std::ratio<3600>>(stop - start).count() << "\n";
    logfile.close();

    start = std::chrono::high_resolution_clock::now();
    std::ofstream out("AgateDistanceMap_fim3.txt", std::ios::out);
    for(int z = 0; z < 360; ++z)
    {
        for(int y = 0; y < 360; ++y)
        {
            for(int x = 0; x < 360; ++x)
            {
                out << x << ' ' << y << ' ' << z << ' ' << arrival_times[x][y][z] << '\n';
            }
        }
    }
    out.close();
    stop = std::chrono::high_resolution_clock::now();
    std::cout << "Writing to file took: " << std::chrono::duration<double>(stop - start).count() <<
              " seconds\n";
}