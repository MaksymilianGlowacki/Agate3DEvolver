#include "hip/hip_runtime.h"
#include "mesh_functions_gpu.cuh"
#include <happly.h> //Lines 997 & 954 dynamic_cast -> static_cast
#include <algorithm>
#include <chrono>

__global__
void calculate_distances(const int X, const int Y, int num_tri, triangle *mesh, double *results)
{
    unsigned int z = threadIdx.x;
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x;
    results[z * (X * Y) + y * X + x] = distance_to_mesh(mesh, x, y, z, num_tri, Y);
}

int main()
{
    happly::PLYData plyIn(R"(C:\Users\Maks\CLionProjects\Agate3DEvolver\AgateContour_0209.ply)");


    std::vector<std::array<double, 3>> vPos = plyIn.getVertexPositions();
    std::vector<std::vector<size_t>> fInd = plyIn.getFaceIndices<size_t>();

    /*
    const int max_x = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[0] < b[0]);}))[0]);
    const int max_y = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[1] < b[1]);}))[1]);
    const int max_z = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[2] < b[2]);}))[2]);
    const int min_x = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[0] < b[0]);}))[0]);
    const int min_y = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[1] < b[1]);}))[1]);
    const int min_z = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[2] < b[2]);}))[2]);
                                                    */
    //const int X = max_x - min_x, Y = max_y - min_y, Z = max_z - min_z;
    const int min_x = 0, min_y = 0, min_z = 0, X = 300, Y = 210, Z = 360;
    triangle *mesh;
    double *results;
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    hipMallocManaged(&mesh, fInd.size() * sizeof(triangle));
    hipMallocManaged(&results, (X * Y * Z) * sizeof(double));

    for(size_t i = 0; i < fInd.size(); ++i)
    {
        mesh[i][0][0] = vPos[fInd[i][0]][0] - min_x;
        mesh[i][0][1] = vPos[fInd[i][0]][1] - min_y;
        mesh[i][0][2] = vPos[fInd[i][0]][2] - min_z + 0.4;

        mesh[i][1][0] = vPos[fInd[i][1]][0] - min_x;
        mesh[i][1][1] = vPos[fInd[i][1]][1] - min_y;
        mesh[i][1][2] = vPos[fInd[i][1]][2] - min_z + 0.4;

        mesh[i][2][0] = vPos[fInd[i][2]][0] - min_x;
        mesh[i][2][1] = vPos[fInd[i][2]][1] - min_y;
        mesh[i][2][2] = vPos[fInd[i][2]][2] - min_z + 0.4;
    }
    std::cout << fInd.size() << " triangles loaded from mesh! Starting calculations... \n";
    auto start = std::chrono::high_resolution_clock::now();
    calculate_distances<<<dim3(X, Y), Z>>>(X, Y, (int)fInd.size(), mesh, results);
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();

    std::ofstream logfile("log.txt", std::ios::app);
    logfile << "GPU " << std::chrono::duration<double, std::ratio<3600>>(stop - start).count() << "\n";
    logfile.close();

    start = std::chrono::high_resolution_clock::now();
    std::ofstream out("AgateDistanceMap_final.txt", std::ios::out);
    for(int z = 0; z < Z; ++z)
    {
        for(int y = 0; y < Y; ++y)
        {
            for(int x = 0; x < X; ++x)
            {
                out << x << ' ' << y << ' ' << z << ' ' << results[z * (X * Y) + y * X + x] << '\n';
            }
        }
    }
    hipFree(mesh);
    hipFree(results);
    out.close();
    stop = std::chrono::high_resolution_clock::now();
    std::cout << "Writing to file took: " << std::chrono::duration<double>(stop - start).count() <<
              " seconds\n";
    return 0;
}
