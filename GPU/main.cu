#include "hip/hip_runtime.h"
#include "mesh_functions_gpu.cuh"
#include <happly.h> //edited lines 954 & 997: dynamic_cast converted to static_cast (avoid compilation error)
#include <algorithm>
#include <chrono>

__global__
void calculate_distances(const int X, const int Y, size_t num_tri, triangle *mesh, double *results)
{
    unsigned int z = threadIdx.x;
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x;
    results[z * (X * Y) + y * X + x] = distance_to_mesh(mesh, x, y, z, num_tri);
}

int main()
{
    happly::PLYData plyIn(R"(C:\Users\Maks\CLionProjects\Agate3DEvolver\AgateContour_1008.ply)");
    std::vector<std::array<double, 3>> vPos = plyIn.getVertexPositions();
    std::vector<std::vector<size_t>> fInd = plyIn.getFaceIndices<size_t>();

    const int max_x = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[0] < b[0]);}))[0]);
    const int max_y = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[1] < b[1]);}))[1]);
    const int max_z = std::ceil((*std::max_element(vPos.begin(), vPos.end(),
                                                   [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                   {return (a[2] < b[2]);}))[2]);
    const int min_x = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[0] < b[0]);}))[0]);
    const int min_y = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[1] < b[1]);}))[1]);
    const int min_z = std::floor((*std::min_element(vPos.begin(), vPos.end(),
                                                    [](const std::array<double, 3>& a, const std::array<double, 3>& b)
                                                    {return (a[2] < b[2]);}))[2]);
    const int X = max_x - min_x, Y = max_y - min_y, Z = max_z - min_z;
    triangle *mesh;
    double *results;
    hipMallocManaged(&mesh, fInd.size()*sizeof(triangle));
    hipMallocManaged(&results, (X * Y * Z)*sizeof(double));

    for(size_t i = 0; i < fInd.size(); ++i)
    {
        mesh[i][0][0] = vPos[fInd[i][0]][0] - min_x;
        mesh[i][0][1] = vPos[fInd[i][0]][1] - min_y;
        mesh[i][0][2] = vPos[fInd[i][0]][2] - min_z;

        mesh[i][1][0] = vPos[fInd[i][1]][0] - min_x;
        mesh[i][1][1] = vPos[fInd[i][1]][1] - min_y;
        mesh[i][1][2] = vPos[fInd[i][1]][2] - min_z;

        mesh[i][2][0] = vPos[fInd[i][2]][0] - min_x;
        mesh[i][2][1] = vPos[fInd[i][2]][1] - min_y;
        mesh[i][2][2] = vPos[fInd[i][2]][2] - min_z;
    }
    std::cout << fInd.size() << " triangles loaded from mesh! Starting calculations... \n";
    auto start = std::chrono::high_resolution_clock::now();

    calculate_distances<<<dim3(X, Y), Z>>>(X, Y, fInd.size(), mesh, results);

    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    std::cout << "Calculations took: " << std::chrono::duration<double, std::ratio<3600>>(stop - start).count() <<
              " hours\nSaving results to .txt file...\n";
    start = std::chrono::high_resolution_clock::now();
    std::ofstream out("AgateDistanceMap3.txt", std::ios::out);
    for(int z = 0; z < Z; ++z)
    {
        for(int y = 0; y < Y; ++y)
        {
            for(int x = 0; x < X; ++x)
            {
                out << x << ' ' << y << ' ' << z << ' ' << results[z * (X * Y) + y * X + x] << '\n';
            }
        }
    }
    hipFree(mesh);
    hipFree(results);
    out.close();
    stop = std::chrono::high_resolution_clock::now();
    std::cout << "Writing to file took: " << std::chrono::duration<double>(stop - start).count() <<
              " seconds\n";
    return 0;
}
