#include "hip/hip_runtime.h"
#include "mesh_functions_gpu.cuh"

__device__
double distance_to_triangle(triangle t, double px, double py, double pz, size_t& inter)
{
    double a, b;
    double d1 = ((t[0][0] - px) * (t[0][0] - px) +
                 (t[0][1] - py) * (t[0][1] - py) +
                 (t[0][2] - pz) * (t[0][2] - pz));
    double d2 = ((t[1][0] - px) * (t[1][0] - px) +
                 (t[1][1] - py) * (t[1][1] - py) +
                 (t[1][2] - pz) * (t[1][2] - pz));
    if(d2 < d1) d1 = d2;
    d2 = ((t[2][0] - px) * (t[2][0] - px) +
          (t[2][1] - py) * (t[2][1] - py) +
          (t[2][2] - pz) * (t[2][2] - pz));
    if(d2 < d1) d1 = d2;

    const double    x1 = t[0][0] - px, y1 = t[0][1] - py, z1 = t[0][2] - pz,
                    x2 = t[1][0] - t[0][0], y2 = t[1][1] - t[0][1], z2 = t[1][2] - t[0][2],
                    x3 = t[2][0] - t[0][0], y3 = t[2][1] - t[0][1], z3 = t[2][2] - t[0][2];

    const double    a_inter = -(x3 * y1 - x1 * y3) / (x3 * y2 - x2 * y3),
                    b_inter = (x2 * y1 - x1 * y2) / (x3 * y2 - x2 * y3),
                    x_inter = (x3 * y2 * z1 - x2 * y3 * z1 - x3 * y1 * z2 + x1 * y3 * z2 + x2 * y1 * z3 - x1 * y2 * z3)/
                                (x3 * y2 - x2 * y3);
     if(a_inter > 0 && b_inter > 0 && (a_inter + b_inter) < 1 && x_inter > 0) ++inter;

    const double d = x3 * x3 * (y2 * y2 + z2 * z2) + (y3 * z2 - y2 * z3) * (y3 * z2 - y2 * z3) -
                     2 * x2 * x3 * (y2 * y3 + z2 * z3) + x2 * x2 * (y3 * y3 + z3 * z3);

    a = -((x3 * x3 * (y1 * y2 + z1 * z2) + (y3 * z1 - y1 * z3) * (y3 * z2 - y2 * z3) -
           x3 * (x2 * y1 * y3 + x1 * y2 * y3 + x2 * z1 * z3 + x1 * z2 * z3) +
           x1 * x2 * (y3 * y3 + z3 * z3)) / d);

    if(a >= 0 && a <= 1)
    {
        b = -((x1 * x3 * (y2 * y2 + z2 * z2) + (y2 * z1 - y1 * z2) * (-y3 * z2 + y2 * z3) +
               x2 * x2 * (y1 * y3 + z1 * z3) - x2 * (x3 * y1 * y2 + x1 * y2 * y3 + x3 * z1 * z2 +
                                                     x1 * z2 * z3)) / d);
        if(b >= 0 && (a + b) <= 1)
        {
            return std::sqrt((x1 + x2 * a + x3 * b) * (x1 + x2 * a + x3 * b) +
                             (y1 + y2 * a + y3 * b) * (y1 + y2 * a + y3 * b) +
                             (z1 + z2 * a + z3 * b) * (z1 + z2 * a + z3 * b));
        }
    }

    a = -((x1 * x2 + y1 * y2 + z1 * z2) / (x2 * x2 + y2 * y2 + z2 * z2));
    if(a >= 0 && a <= 1)
    {
        d2 =    (x1 + x2 * a) * (x1 + x2 * a) +
                (y1 + y2 * a) * (y1 + y2 * a) +
                (z1 + z2 * a) * (z1 + z2 * a);
        if(d2 < d1) d1 = d2;
    }

    b = -((x1 * x3 + y1 * y3 + z1 * z3) / (x3 * x3 + y3 * y3 + z3 * z3));
    if(b >= 0 && b <= 1)
    {
        d2 =    (x1 + x3 * b) * (x1 + x3 * b) +
                (y1 + y3 * b) * (y1 + y3 * b) +
                (z1 + z3 * b) * (z1 + z3 * b);
        if(d2 < d1) d1 = d2;
    }

    a = -(((x2 - x3) * (x1 + x3) + (y2 - y3) * (y1 + y3) + (z2 - z3) *
            (z1 + z3)) / ((x2 - x3) * (x2 - x3) + (y2 - y3) * (y2 - y3) + (z2 - z3) * (z2 - z3)));
    if(a >= 0 && a <= 1)
    {
        b = 1 - a;
        d2 =    (x1 + x2 * a + x3 * b) * (x1 + x2 * a + x3 * b) +
                (y1 + y2 * a + y3 * b) * (y1 + y2 * a + y3 * b) +
                (z1 + z2 * a + z3 * b) * (z1 + z2 * a + z3 * b);
        if(d2 < d1) d1 = d2;
    }
    return std::sqrt(d1);
}

__device__
double distance_to_mesh(triangle *mesh, unsigned int x, unsigned int y, unsigned int z, size_t mesh_size)
{
    size_t intersected = 0;
    double d_min = distance_to_triangle(mesh[0], x, y, z, intersected), d;
    for(size_t i = 1; i < mesh_size; ++i)
    {
        d = distance_to_triangle(mesh[i], x, y, z, intersected);
        if(d < d_min) d_min = d;
    }
    return (intersected % 2 == 0) ? d_min : -d_min;
}